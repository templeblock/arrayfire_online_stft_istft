#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#define D_SCL_SECURE_NO_WARNINGS

#include <arrayfire.h>
#include "cpp_buffer_to_wave/WaveManager_revision\WaveManager.h"
#include "complex_double_binary_to_matlab\binary2matlab_converter.hpp"

// var. for test 

typedef af::array arr;
typedef af::dim4 dim4;
typedef af::seq seq;

class stft_AF
{
    typedef af::array arr;
    typedef af::dim4 dim4;
    typedef af::seq seq;

    int sfft;
    int nch;
    int sshift;
    int nfreq;
    
    arr window;
    arr outWindow;

    std::vector<double> vHanHalfWinCoeff;
    arr hanHalfWinCoeff;

    arr hanWinCoeff;

public:

    stft_AF(const int sfft, const int nch, const int sshift)
    {
        // make hanning window
        initialize(sfft, nch, sshift);
    }

    void initialize(const int _sfft, const int _nch, const int _sshift)
    {
        sfft = _sfft;
        sshift = _sshift;
        nfreq = _sfft / 2 + 1;
        nch = _nch;

        window = af::constant(0.0f, sfft, nch, c32);
        outWindow = af::constant(0.0f, sfft, nch, f32);

        make_hanning_window();
    }

    void make_hanning_window()
    {
        std::cout << "make_hanning_window\n";

        vHanHalfWinCoeff.resize(sfft/2+1);
        
        for (int sample = 0; sample < sfft/2 + 1; sample++)
            vHanHalfWinCoeff[sample] = 0.5 * (1.0 - cos(2.0 * 3.14159265358979323846*(double)(sample) / ((double)sfft)));

        hanWinCoeff = af::constant(0.0f, sfft, c32);
        hanHalfWinCoeff = arr(sfft/2+1, &vHanHalfWinCoeff[0]).as(c32);

        hanWinCoeff(seq(0, sfft / 2)) = hanHalfWinCoeff;
        hanWinCoeff(seq(sfft / 2 + 1, sfft - 1, 1)) = hanHalfWinCoeff(seq(sfft / 2 - 1, 1, -1));

        hanWinCoeff = sshift*hanWinCoeff / af::tile(af::sum(hanWinCoeff), sfft, 1);
    }

    ~stft_AF()
    {

    }

    // in: time domain windowed. dim: (sshift x nch)
    // out: freq domain frame. dim: (nfreq x nch)
    arr stft(arr& in)
    {
        // shifting
        window(seq(0, (sfft-sshift-1)), af::span) = window(seq(sshift, af::end), af::span);
        
        // copying new samples
        window(seq((sfft - sshift), af::end), af::span) = in;
        arr windowed = af::batchFunc(window, hanWinCoeff, af::operator*);
        
        // batch fft
        arr fftd = af::fft(windowed, sfft);

        // return cropped last (1 + half) -> size: (sfft / 2 + 1)
        return fftd(seq(0, nfreq-1), af::span);
    }

    // in: freq domain frame. dim: (nfreq x nch)
    // out: time domain windowed. dim: (sshift x nch)
    af::array istft(af::array& frame, const bool isEnd)
    {
        
        //shifting
        outWindow(seq(0, af::end - sshift), af::span) = outWindow(seq(sshift, af::end), af::span);
        
        outWindow(seq(af::end - sshift, af::end), af::span) = 0.0f;
        
        //adding
        outWindow = outWindow + af::real(ifft(af::join(0, frame, af::conjg(frame(seq(af::end - 1, 1, -1), af::span)))));
        
        return outWindow(seq(0, sshift - 1), af::span);
    }

};



int main(void)
{
    const int sfft = 512;
    const int nfreq = sfft / 2+1;
    const int soverlap = sfft / 2;
    const int sshift = sfft - soverlap;
    const int nCh = 7;

    stft_AF* pstft = new stft_AF(sfft, nCh, sshift);

    float** buffer;
    int nBlocks = wm::wread("wav_7ch.wav", buffer);

    int nbuffers = nBlocks / sshift;

    arr samples = af::constant(0.0f, nBlocks, nCh, c32);

    arr frames = af::constant(0.0f, nfreq, nCh, nbuffers, c32);

    arr processed = af::constant(0.0f, nBlocks, nCh, f32);

    
    for (int i = 0; i < nCh; i++)
    {
        samples(af::span, i) = arr(nBlocks, (&buffer[i][0]), afHost).as(c32);
    }
    
    arr frame = af::constant(0.0f, nfreq, c32);

    af::timer::start();
    for (int i = 0; i < nbuffers; i++)
    {
        int start = i * sshift;
        
        arr arrived = samples(seq(start, start + sshift - 1), af::span);
        
        frame = pstft->stft(arrived);
        frames(af::span, af::span, i) = frame;
        processed(seq(start, start + sshift - 1), af::span) = pstft->istft(frame, false);
    }
    printf("elapsed seconds per 1 frame: %g\n", af::timer::stop() / (double)nbuffers);

    float** ptr7ch = new float*[nCh];

    for (int i = 0; i < nCh; i++)
        ptr7ch[i] = (processed(af::span, i)).host<float>();

    wm::wwrite("ch7_processed.wav", ptr7ch, nBlocks, nCh);

    processed.unlock();

    delete pstft;

    delete buffer;

    for (int i = 0; i < nCh; i++)
        delete ptr7ch[i];

    delete ptr7ch;

    return 0;
}