#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#define D_SCL_SECURE_NO_WARNINGS

#include <arrayfire.h>
#include "cpp_buffer_to_wave/WaveManager_revision\WaveManager.h"
#include "complex_double_binary_to_matlab\binary2matlab_converter.hpp"

// var. for test 

typedef af::array arr;
typedef af::dim4 dim4;
typedef af::seq seq;

class stft_AF
{
    typedef af::array arr;
    typedef af::dim4 dim4;
    typedef af::seq seq;

    int sfft;
    int nch;
    int soverlap;
    int sfreq;
    

    arr window;
    
    std::vector<double> vHanWinCoeff;
    arr hanWinCoeff;

public:

    stft_AF(const int sfft, const int nch, const int soverlap)
    {
        // make hanning window
        initialize(sfft, nch, soverlap);
    }

    void initialize(const int _sfft, const int _nch, const int _soverlap)
    {
        sfft = _sfft;
        soverlap = _soverlap;
        sfreq = _sfft / 2 + 1;
        nch = _nch;

        window = af::constant(0.0f, sfft, nch, c32);

        make_hanning_window();
    }

    void make_hanning_window()
    {
        std::cout << "make_hanning_window\n";

        vHanWinCoeff.resize(sfft);
        
        std::cout << "make_hanning_window\n";

        for (int sample = 0; sample < sfft; sample++)
            vHanWinCoeff[sample] = 0.5 * (1.0 - cos(2.0 * 3.14159265358979323846*(double)(sample) / ((double)sfft)));

        std::cout << "make_hanning_window\n";
        hanWinCoeff = arr(sfft, 1, &vHanWinCoeff[0]).as(c32);
    }

    ~stft_AF()
    {

    }

    // in: time domain windowed. dim: (soverlap x nch)
    // out: freq domain frame. dim: (sfreq x nch)
    arr stft(arr& in)
    {
        //std::cout << in.dims() << std::endl;
        //std::cout << window.dims() << std::endl;

        //std::cout << "sfft: " << sfft << std::endl;
        //std::cout << "soverlap: " << soverlap << std::endl;
        //

        //std::cout << "here2\n";
        // shifting
        window(seq(0, (sfft-soverlap-1)), af::span) = window(seq(soverlap, af::end), af::span);
        //std::cout << "here2\n";
        // copying new samples
        window(seq((sfft - soverlap), af::end), af::span) = in;
        //std::cout << "here2\n";        // batch windowing
        arr windowed = af::batchFunc(window, hanWinCoeff, af::operator*);
        //std::cout << "here2\n";
        // batch fft
        arr fftd = af::fft(windowed, sfft);
        //std::cout << "here2\n";
        // return cropped last (1 + half) -> size: (sfft / 2 + 1)
        //arr cropped = fftd(seq(0, sfreq), af::span);

        return fftd(seq(0, sfreq-1), af::span);
    }
};



int main(void)
{
    const int sfft = 512;
    const int sfreq = sfft / 2+1;
    const int soverlap = sfft / 2;

    const int nCh = 7;

    stft_AF* pstft = new stft_AF(sfft, nCh, soverlap);

    float** buffer;
    int nSamples = wm::wread("wav_7ch.wav", buffer);

    int nbuffers = nSamples / soverlap;

    //std::cout << "nbuffers: " << nbuffers << std::endl;

    arr frames = arr(sfreq, nCh, nbuffers, c32);

    arr samples = af::constant(0.0f, nSamples, nCh, c32);
    //std::cout << "herehere\n";
    for (int i = 0; i < nCh; i++)
    {
        samples(af::span, i) = arr(nSamples, (&buffer[i][0]), afHost).as(c32);
    }
    //std::cout << "herehere\n";

    af::timer::start();
    for (int i = 0; i < nbuffers; i++)
    {
        int start = i * soverlap;
        //std::cout << "here\n";
        arr arrived = samples(seq(start, start + soverlap - 1), af::span);
        //std::cout << "here\n";
        frames(af::span, af::span, i) = pstft->stft(arrived);
    }
    printf("elapsed seconds per 1 frame: %g\n", af::timer::stop() / (double)nbuffers);


    //std::cout << "herehere\n";
    af::write_complex64_binary(frames, "stfted");
    

    delete pstft;

    delete buffer;

    return 0;
}